#include "../lib/kernels.hpp"
#include "../lib/timer.h"
#include "../lib/cuda_helper.h"
#include <cstdio>

void eig_dsymx_cusolver(
    hipsolverHandle_t cusolverH, 
    double* d_A, 
    double* d_W,
    int m,
    int k, 
    double* d_eigvals,
    double* d_eigvecs
    )
{
    // Calculates eigenvalues and eigenvectors of a symmetric matrix using cusolver
    // Arguments:
    //      cusolverH: cusolver handle
    //      d_A: matrix to calculate eigenvalues and eigenvectors of
    //      d_W: array to store eigenvalues in
    //      m: size of the matrix
    //      k: number of eigenvalues to calculate
    //      d_eigvals: array to store eigenvalues in
    //      d_eigvecs: array to store eigenvectors in
    if (k == -1) {
        // Raise exception
        printf("eig_dsymx_cusolver: k must be value > 0!\n");                          \
        throw std::runtime_error("eig_dsymx_cusolver");
    }

    void* d_work;
    int* devInfo;
    int h_meig = k; // number of eigenvalues found in the interval
    int workspaceInBytes;

    
    CUSOLVER_CHECK(hipsolverDnDsyevdx_bufferSize(
        cusolverH,
        HIPSOLVER_EIG_MODE_VECTOR, // compute eigenvectors
        HIPSOLVER_EIG_RANGE_I,     // compute eigenvalues in an interval
        HIPBLAS_FILL_MODE_LOWER,
        m,
        d_A,
        m,
        0.0, // vl - not used
        0.0, // vu - not used
        m-k+1,
        m,
        &h_meig,
        d_W,
        &workspaceInBytes
    ));
    
    // Initialize the workspace
    CUDA_CHECK(hipMalloc(&d_work, workspaceInBytes * sizeof(double)));
    CUDA_CHECK(hipMalloc(&devInfo, sizeof(int)));


    CUSOLVER_CHECK(hipsolverDnDsyevdx(
        cusolverH,
        HIPSOLVER_EIG_MODE_VECTOR, // compute eigenvectors.
        HIPSOLVER_EIG_RANGE_I,     // compute eigenvalues in an interval
        HIPBLAS_FILL_MODE_LOWER,
        m,      // size of the matrix
        d_A,    // matrix
        m,      // leading dimension of A
        0.0,    // vl - not used
        0.0,    // vu - not used
        m-k+1,    // il - lower bound of interval (index)
        m,      // iu - upper bound of interval (index)
        &h_meig,// number of eigenvalues found in the interval
        d_W,    // eigenvalues
        (double*)d_work,    // workspace
        workspaceInBytes,
        devInfo // error info
    ));
    

    // Initialize the eigenvalues and eigenvectors
    if (d_eigvals != nullptr)
        CUDA_CHECK(hipMemcpy(d_eigvals, d_W, sizeof(double) * k, hipMemcpyDeviceToDevice));
    if (d_eigvecs != nullptr)
        CUDA_CHECK(hipMemcpy(d_eigvecs, d_A, sizeof(double) * m * k, hipMemcpyDeviceToDevice));

    CUDA_CHECK(hipFree(d_work));
    CUDA_CHECK(hipFree(devInfo));
}

void test_eig_kernel(arma::mat mat, int  m)
{
    hipsolverHandle_t cusolverH;
    hipsolverStatus_t cusolver_status;
    hipsolverDnCreate(&cusolverH);
    double* d_A;
    double* d_W;
    void* d_work;
    int* devInfo;
    int h_meig; // number of eigenvalues found in the interval
    int workspaceInBytes;

    CUDA_CHECK(hipMalloc(&d_A, sizeof(double) * m * m)); // Won't be needed if we use the matrix directly in the argument

    cusolver_status = hipsolverDnDsyevdx_bufferSize(
        cusolverH,
        HIPSOLVER_EIG_MODE_VECTOR, // compute eigenvectors
        HIPSOLVER_EIG_RANGE_I,     // compute eigenvalues in an interval
        HIPBLAS_FILL_MODE_LOWER,
        m,
        d_A,
        m,
        0.0, // vl - not used
        0.0, // vu - not used
        m-1,
        m,
        &h_meig,
        d_W,
        &workspaceInBytes
    );
    // hipMalloc(&d_W, m * sizeof(double));
    hipMalloc(&d_work, workspaceInBytes * sizeof(double));
    hipMalloc(&devInfo, sizeof(int));
    printf("Workspace size: %d\n", workspaceInBytes);
    int workspaceInBytes2;
    cusolver_status = hipsolverDnDsyevd_bufferSize(
            cusolverH,
            HIPSOLVER_EIG_MODE_VECTOR, // compute eigenvectors
            HIPBLAS_FILL_MODE_LOWER,
            m,
            d_A,
            m,
            d_W,
            &workspaceInBytes2
    );
    printf("Workspace size: %d\n", workspaceInBytes2);

    hipMemcpy(d_A, mat.memptr(), sizeof(double) * m * m, hipMemcpyHostToDevice);

    cusolver_status = hipsolverDnDsyevdx(
        cusolverH,
        HIPSOLVER_EIG_MODE_VECTOR, // compute eigenvectors.
        HIPSOLVER_EIG_RANGE_I,     // compute eigenvalues in an interval
        HIPBLAS_FILL_MODE_LOWER,
        m,      // size of the matrix
        d_A, 
        m,      // leading dimension of A
        0.0,    // vl - not used
        0.0,    // vu - not used
        m-1,    // il - lower bound of interval (index)
        m,      // iu - upper bound of interval (index)
        &h_meig,// number of eigenvalues found in the interval
        d_W,    // eigenvalues
        (double*)d_work,    // workspace
        workspaceInBytes,
        devInfo // error info
    );

    // cusolver_status = hipsolverDnDsyevd(
    //         cusolverH,
    //         HIPSOLVER_EIG_MODE_VECTOR, // compute eigenvectors.
    //         HIPBLAS_FILL_MODE_LOWER,
    //         m,      // size of the matrix
    //         d_A,    // matrix
    //         m,      // leading dimension of A
    //         d_W,    // eigenvalues
    //         (double*)d_work,    // workspace
    //         workspaceInBytes,
    //         devInfo // error info
    //     );

    // Check dev info
    int devInfo_h = 0;
    hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    printf("devInfo = %d\n", devInfo_h);
    arma::vec eigenvalues(m);
    hipMemcpy(eigenvalues.memptr(), d_W, sizeof(double) * 2, hipMemcpyDeviceToHost);
    eigenvalues.save("eigenvalues.txt", arma::raw_ascii);
    hipMemcpy(mat.memptr(), d_A, sizeof(double) * m * m, hipMemcpyDeviceToHost);
    mat.save("eigenvectors.txt", arma::raw_ascii);
    printf("h_meig = %d\n", h_meig);

    hipFree(d_A);
    hipFree(d_W);
    hipFree(d_work);
    hipFree(devInfo);
    hipsolverDnDestroy(cusolverH);
}